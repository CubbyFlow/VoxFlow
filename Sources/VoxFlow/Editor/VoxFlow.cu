#include "hip/hip_runtime.h"
// Author : snowapril

#include <hip/hip_runtime.h>
#include <VoxFlow/Editor/VoxFlow.hpp>

namespace VoxFlow
{
VoxFlow::VoxFlow()
{
}
VoxFlow::~VoxFlow()
{
}

bool VoxFlow::initialize()
{
    auto solver = CUDAPCISPHSolver3::Builder().MakeShared();
    solver->SetDragCoefficient(0.0f);
    solver->SetRestitutionCoefficient(0.0f);
    solver->SetViscosityCoefficient(0.1f);
    solver->SetPseudoViscosityCoefficient(10.0f);
    solver->SetIsUsingFixedSubTimeSteps(true);
    solver->SetNumberOfFixedSubTimeSteps(1);

    auto particles = solver->SPHSystemData();
    particles->SetTargetSpacing(targetSpacing);
    particles->SetRelativeKernelRadius(1.8f);

    // Seed particles
    BoundingBox3D vol(Vector3D(), Vector3D(0.5, 0.5, 0.5));
    vol.Expand(-targetSpacing);
    Array1<Vector3D> rawPoints;
    GridPointGenerator3 generator;
    generator.Generate(vol, targetSpacing, &rawPoints);
    Array1<float4> hostData(rawPoints.Length());
    for (std::size_t i = 0; i < rawPoints.Length(); ++i)
    {
        Vector3F rp = rawPoints[i].CastTo<float>();
        hostData[i] = make_float4(rp[0], rp[1], rp[2], 0.0f);
    }
    CUDAArray1<float4> deviceData(hostData);
    particles->AddParticles(deviceData);
    printf("Number of particles: %zu\n", particles->NumberOfParticles());
}

void VoxFlow::updateSolver()
{
    for (Frame frame(0, 1.0 / 60.0); frame.index < 10; ++frame)
    {
        solver->Update(frame);
    }
}
}  // namespace VoxFlow

#endif